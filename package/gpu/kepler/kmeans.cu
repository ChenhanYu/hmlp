#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/tuple.h>

#include<hmlp.h>
#include<hmlp_blas_lapack.h>
#include<gkmx_gpu.hpp>

using namespace hmlp;


template <typename TV, typename TC>
struct kmeans 
{
  __host__ __device__ __forceinline__ TC operator()
  ( 
    const TV& x, int i, int j, int b 
  ) const 
  { 
    return thrust::make_pair( x , j ); 
  }
  TV** A2;
  TV** B2;
};

template <typename TC>
struct argmin 
{
  __host__ __device__ __forceinline__ TC operator()
  ( 
    const TC& lhs, const TC& rhs, int i, int j, int b 
  ) const 
  {
    return lhs.first < rhs.first ? lhs : rhs;
  }
};




template<typename T>
void kmeans_ref
(
  hipStream_t stream, 
  //hmlpOperation_t transA, hmlpOperation_t transB, 
  int m, int n, int k,
  T *Aarray[], T *A2array[], int lda,
  T *Barray[], T *B2array[], int ldb,
  thrust::pair<T,int>  *Carray[], int ldc, 
  int batchSize
)
{
  using TC = thrust::pair<T, int>;

  hipblasHandle_t handle;
  hipblasCreate( &handle );

  printf( "m %d ldc %d n %d k %d batchSize %d\n", m, ldc, n, k, batchSize );

  thrust::device_vector<T>  Varray( ldc * n * batchSize, 0.0 );
  thrust::device_vector<T*> Varrayp( batchSize );


  printf( "after allocate\n" );

  kmeans<T, TC> opkernel;
  argmin<TC> opreduce;

  // Declare <TC> initial value.
  TC initC( 999999.99, -1 );

  opkernel.A2 = A2array;
  opkernel.B2 = B2array;


  printf( "after thrust::device\n" );

  for ( int i = 0; i < batchSize; i ++ ) 
  {
    Varrayp[ i ] = Varray.data().get() + i * ldc * n;
  }

  xgemm_batched
  (
    handle,
    HIPBLAS_OP_T, HIPBLAS_OP_N,
    m, n, k,
    1.0,
    Aarray, lda,
    Barray, ldb, 0.0,
    Varrayp.data().get(), ldc,
    batchSize
  );


  printf( "after xgemm_batched\n" );

  // Compute the 2-norm here and reduce
  gkmx::transform
  <T, TC, false, true, kmeans<T, TC> >
  (
    0,
    m, n, 
    Varrayp.data().get(), (T*)NULL, 
    Carray, (TC*)NULL, ldc, ldc * n,
    batchSize, 
    opkernel
  )
  ;
  printf( "after transform\n" );


  gkmx::reduce
  <TC, false, argmin<TC> >
  (
    0,
    m, n,
    Carray, Carray[ 0 ], ldc, ldc * n,
    batchSize,
    opreduce, initC
  );
};

void dkmeans
(
  hipStream_t stream, 
  //hmlpOperation_t transA, hmlpOperation_t transB, 
  int m, int n, int k,
  double *Aarray[], double *A2array[], int lda,
  double *Barray[], double *B2array[], int ldb,
  thrust::pair<double,int>  *Carray[], int ldc, 
  int batchSize
)
{
  kmeans_ref<double>
  (
    stream,
    m, n, k,
    Aarray, A2array, lda,
    Barray, B2array, ldb,
    Carray,          ldc,
    batchSize
  );
}

void skmeans
(
  hipStream_t stream, 
  //hmlpOperation_t transA, hmlpOperation_t transB, 
  int m, int n, int k,
  float *Aarray[], float *A2array[], int lda,
  float *Barray[], float *B2array[], int ldb,
  thrust::pair<float,int>  *Carray[], int ldc, 
  int batchSize
)
{
  kmeans_ref<float>
  (
    stream,
    m, n, k,
    Aarray, A2array, lda,
    Barray, B2array, ldb,
    Carray,          ldc,
    batchSize
  );
}
