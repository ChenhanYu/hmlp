#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include <hmlp_blas_lapack.h>

namespace hmlp
{

void xgemm_batched
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *Aarray[], int lda,
  double *Barray[], int ldb, double beta,
  double *Carray[], int ldc,
  int batchSize
)
{
  hipblasDgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double**)Aarray, lda,
    (const double**)Barray, ldb, &beta,
                    Carray, ldc,
    batchSize
  );
};

}; // end namespace hmlp
