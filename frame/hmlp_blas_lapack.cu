#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include <hmlp_blas_lapack.h>


namespace hmlp
{

  
void xgemm_batched
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *Aarray[], int lda,
  double *Barray[], int ldb, double beta,
  double *Carray[], int ldc,
  int batchSize
)
{
  hipblasDgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double**)Aarray, lda,
    (const double**)Barray, ldb, &beta,
                    Carray, ldc,
    batchSize
  );
};


void xgemm_batched
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  float alpha,
  float *Aarray[], int lda,
  float *Barray[], int ldb, float beta,
  float *Carray[], int ldc,
  int batchSize
)
{
  hipblasSgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const float**)Aarray, lda,
    (const float**)Barray, ldb, &beta,
                   Carray, ldc,
    batchSize
  );
};



}; // end namespace hmlp
