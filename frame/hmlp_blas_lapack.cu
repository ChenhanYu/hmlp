#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include <hmlp_blas_lapack.h>


namespace hmlp
{

// hipblasDgemm wrapper
void xgemm
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *A, int lda,
  double *B, int ldb, double beta,
  double *C, int ldc
)
{
  hipblasDgemm
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double*)A, lda,
    (const double*)B, ldb, &beta,
                   C, ldc
  );
};

// hipblasSgemm wrapper
void xgemm
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  float alpha,
  float *A, int lda,
  float *B, int ldb, float beta,
  float *C, int ldc
)
{
  hipblasSgemm
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const float*)A, lda,
    (const float*)B, ldb, &beta,
                   C, ldc
  );
};


// hipblasDgemmBatched wrapper
void xgemm_batched
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *Aarray[], int lda,
  double *Barray[], int ldb, double beta,
  double *Carray[], int ldc,
  int batchSize
)
{
  hipblasDgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double**)Aarray, lda,
    (const double**)Barray, ldb, &beta,
                    Carray, ldc,
    batchSize
  );
};


// hipblasSgemmBatched wrapper
void xgemm_batched
(
  hipblasHandle_t handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  float alpha,
  float *Aarray[], int lda,
  float *Barray[], int ldb, float beta,
  float *Carray[], int ldc,
  int batchSize
)
{
  hipblasSgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const float**)Aarray, lda,
    (const float**)Barray, ldb, &beta,
                   Carray, ldc,
    batchSize
  );
};



}; // end namespace hmlp
