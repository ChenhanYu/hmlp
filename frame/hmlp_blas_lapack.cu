#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include <hmlp_blas_lapack.h>

#ifdef HMLP_USE_MAGMA
#include <magma_v2.h>
#include <magma_lapack.h>
#endif

namespace hmlp
{

// hipblasDgemm wrapper
void xgemm
(
  hipblasHandle_t &handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *A, int lda,
  double *B, int ldb, double beta,
  double *C, int ldc
)
{
  hipblasDgemm
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double*)A, lda,
    (const double*)B, ldb, &beta,
                   C, ldc
  );
};

// hipblasSgemm wrapper
void xgemm
(
  hipblasHandle_t &handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  float alpha,
  float *A, int lda,
  float *B, int ldb, float beta,
  float *C, int ldc
)
{
  hipblasSgemm
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const float*)A, lda,
    (const float*)B, ldb, &beta,
                   C, ldc
  );
};


// hipblasDgemmBatched wrapper
void xgemm_batched
(
  hipblasHandle_t &handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  double alpha,
  double *Aarray[], int lda,
  double *Barray[], int ldb, double beta,
  double *Carray[], int ldc,
  int batchSize
)
{
  hipblasDgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const double**)Aarray, lda,
    (const double**)Barray, ldb, &beta,
                    Carray, ldc,
    batchSize
  );
};


// hipblasSgemmBatched wrapper
void xgemm_batched
(
  hipblasHandle_t &handle,
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k, 
  float alpha,
  float *Aarray[], int lda,
  float *Barray[], int ldb, float beta,
  float *Carray[], int ldc,
  int batchSize
)
{
  hipblasSgemmBatched
  (
    handle,
    transA, transB,
    m, n, k,
    &alpha,
    (const float**)Aarray, lda,
    (const float**)Barray, ldb, &beta,
                   Carray, ldc,
    batchSize
  );
};

// magma_dgeqp3 wrapper
void xgeqp3
(
  hipblasHandle_t &handle,
  int m, int n,
  double *A, int lda,
  int *jpvt,
  double *tau,
  double *work, int lwork
)
{
#ifdef HMLP_USE_MAGMA
  printf( "magma_dgeqp3\n" );
  int info = 0;
  magma_dgeqp3
  (
    m, n, 
    A, lda,
    jpvt,
    tau,
    work, lwork,
    &info
  );
#else
  xgeqp3
  (
    m, n, 
    A, lda,
    jpvt,
    tau,
    work, lwork
  );
#endif
};


// magma_sgeqp3 wrapper
void xgeqp3
(
  hipblasHandle_t &handle,
  int m, int n,
  float *A, int lda,
  int *jpvt,
  float *tau,
  float *work, int lwork
)
{
#ifdef HMLP_USE_MAGMA
  printf( "magma_sgeqp3\n" );
  int info = 0;
  magma_sgeqp3
  (
    m, n, 
    A, lda,
    jpvt,
    tau,
    work, lwork,
    &info
  );
#else
  xgeqp3
  (
    m, n, 
    A, lda,
    jpvt,
    tau,
    work, lwork
  );
#endif
};





}; // end namespace hmlp
